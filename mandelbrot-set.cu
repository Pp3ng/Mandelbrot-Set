#include <iostream>
#include <fstream>
#include <cmath>
#include <hip/hip_runtime.h>

const int width = 8192;
const int height = 8192;
const int max_iterations = 8000;

__device__ int mandelbrot(float real, float imag)
{
    float r = real;
    float i = imag;
    for (int iter = 0; iter < max_iterations; ++iter)
    {
        float r2 = r * r;
        float i2 = i * i;
        if (r2 + i2 > 4.0f)
        {
            return iter;
        }
        i = 2.0f * r * i + imag;
        r = r2 - i2 + real;
    }
    return max_iterations;
}

__global__ void generateMandelbrotSet(int *output)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float real = (x - width / 2.0) * 4.0 / width;
    float imag = (y - height / 2.0) * 4.0 / height;

    int value = mandelbrot(real, imag);

    output[y * width + x] = value;
}

int main()
{
    int *host_output = new int[width * height];
    int *device_output;

    hipMalloc((void **)&device_output, width * height * sizeof(int));

    dim3 blockDim(32, 32);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    generateMandelbrotSet<<<gridDim, blockDim>>>(device_output);

    hipMemcpy(host_output, device_output, width * height * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_output);

    std::ofstream image("mandelbrot_set_cuda.ppm");
    image << "P3\n"
          << width << " " << height << "\n255\n";

    for (int i = 0; i < width * height; ++i)
    {
        int value = host_output[i];
        int r = 0;
        int g = 0;
        int b = (value % 256);
        image << r << " " << g << " " << b << "\n";
    }

    delete[] host_output;

    return 0;
}
